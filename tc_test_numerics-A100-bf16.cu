/*
 * Copyright (c) 2020, Massimiliano Fasi and Mantas Mikaitis
 *
 * This program is free software: you can redistribute it and/or modify it under
 * the terms of the GNU General Public License as published by the Free Software
 * Foundation, version 2.
 *
 * This program is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE. See the GNU General Public License for more
 * details.
 *
 *  You should have received a copy of the GNU General Public License along with
 *  this program. If not, see <http://www.gnu.org/licenses/>.
 */


#include <hip/hip_runtime.h>
#include <assert.h>
#include <unistd.h>
#include <cstdint>
#include <chrono>
#include <iostream>
#include <mma.h>
#include <iomanip>
#include <hip/hip_bf16.h>

using namespace nvcuda;

/*******************
 * Debug functions *
 *******************/
/* Print the elements of the m x n matrix A. The elements are assumed to be
   stored by columns if `bycols` is `true` and by rows if `bycols` is false. */
template <typename floattype>
void print_matrix (hip_bfloat16 *a,
                   size_t m, size_t n,
                   bool bycols) {
  int i, j;
  if (bycols) {
    for (i=0; i<m; i++) {
      for (j=0; j<n; j++)
        std::cout << __bfloat162float(a[j*n+i]) << " ";
      std::cout << std::endl;
    }
    std::cout << std::endl;
  } else {
    for (i=0; i<m; i++ ) {
      for (j=0; j<n; j++)
        std::cout << __bfloat162float(a[i*m+j]) << " ";
      std::cout  << std::endl;
    }
    std::cout << std::endl;
   }
}


/****************************************************
 * Memory management and wmma::mma_sync() interface *
 ****************************************************/

/* Set the entries of host arrays to zero. */
template <typename returntype>
void host_reset(hip_bfloat16 *a, hip_bfloat16 *b, returntype *c) {
  memset(a, 0, 16*16*sizeof(hip_bfloat16));
  memset(b, 0, 16*16*sizeof(hip_bfloat16));
  memset(c, 0, 16*16*sizeof(returntype));
}

/* Compute C += A*B, where A, B, and C are 16x16x16 matrices.
   The matrix C is initialized to 0 when `init` is true. */
template <typename returntype>
__global__ void wmma_ker(hip_bfloat16 *a, hip_bfloat16 *b,
                         returntype *c, bool init) {

  // Declare fragments.
  wmma::fragment<wmma::matrix_a, 16, 16, 16, hip_bfloat16,
    wmma::row_major> a_fragment;
  wmma::fragment<wmma::matrix_b, 16, 16, 16, hip_bfloat16,
    wmma::col_major> b_fragment;
  wmma::fragment<wmma::accumulator, 16, 16, 16, returntype> c_fragment;

  // Load input matrices and initialize output (if required).
  wmma::load_matrix_sync(a_fragment, a, 16);
  wmma::load_matrix_sync(b_fragment, b, 16);
  if (init)
    wmma::fill_fragment(c_fragment, 0.0f);
  else
    wmma::load_matrix_sync(c_fragment, c, 16, wmma::mem_col_major);

  // Multiply
  wmma::mma_sync(c_fragment, a_fragment, b_fragment, c_fragment);

  // Store the output
  wmma::store_matrix_sync(c, c_fragment, 16, wmma::mem_col_major);
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/* Copy data from host to device, perform the operation, and copy result back to
   host. */
template <typename returntype>
void wmma_init_run (hip_bfloat16 *h_a, hip_bfloat16 *h_b, returntype *h_c,
                    hip_bfloat16 *d_a, hip_bfloat16 *d_b, returntype *d_c,
                    bool init) {
  
  gpuErrchk( (hipGetLastError()) );

  // Copy input from host to device.
  hipMemcpy(d_a, h_a, 16*16*sizeof(hip_bfloat16), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, 16*16*sizeof(hip_bfloat16), hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, 16*16*sizeof(returntype), hipMemcpyHostToDevice);

  gpuErrchk( (hipGetLastError()) );

  // Perform matrix multiplication.
  wmma_ker<<<1,32>>>(d_a, d_b, d_c, init);

  gpuErrchk( (hipGetLastError()) );

  // Copy result from device to host.
  hipMemcpy(h_c, d_c, 16*16*sizeof(returntype), hipMemcpyDeviceToHost);

  gpuErrchk( (hipGetLastError()) );
}


/**********************
 * Printing functions *
 **********************/
void printheader(FILE *outfile, const char *string) {
  fprintf(outfile,
          "+--------------------------------------------------------------+\n");
  fprintf(outfile, "| %-60s |\n", string);
  fprintf(outfile,
          "+--------------------------------------------------------------+\n");
}
void printitem(FILE *outfile, const char *string) {
  fprintf(outfile, "  | %-49s", string);
}

void printpass(FILE *outfile, bool status) {
  if (status)
    fprintf(outfile, " [PASS] |\n");
  else
    fprintf(outfile, " [FAIL] |\n");
}
void printfooter(FILE *outfile) {
  fprintf(outfile,
          "  +----------------------------------------------------------+\n\n");
}


/***************
 * EXPERIMENTS *
 ***************/
int main(int argc, char** argv){

  // Declare pointers and allocate memory.
  hip_bfloat16 *h_a, *h_b, *h16_c, *d16_a, *d16_b, *d16_c,
    minsubnormal16 = __float2bfloat16(ldexp(1., -133)), // smallest subn. bf16
    belowone16 = __float2bfloat16(1. - ldexp(1, -8)),
    zero16 = __float2bfloat16(0.),
    one16 = __float2bfloat16(1.),
    minusone16 = __float2bfloat16(-1.),
    two16 = __float2bfloat16(2.),
    four16 = __float2bfloat16(4.);
  float *d_c, *h_c,
    minsubnormal32 = ldexp(1., -149), // smallest subnormal binary32
    belowone = nextafterf(1., 0.) ,   // largest float smaller than 1.0
    gapbelowone = 1. - belowone,
    aboveone = nextafterf(1., 2.),    // smallest float larger than 1.0
    belowtwo = 2. - ldexp(1., -23);   // largest float smaller than 2.0

  assert(belowone == 1. - ldexp(1., -24));
  assert(aboveone == 1. + ldexp(1., -23));

  h_a = new hip_bfloat16[16*16];
  h_b = new hip_bfloat16[16*16];
  h_c = new float[16*16];
  h16_c = new hip_bfloat16[16*16];

  hipMalloc(&d16_a, 16*16*sizeof(hip_bfloat16));
  hipMalloc(&d16_b, 16*16*sizeof(hip_bfloat16));
  hipMalloc(&d16_c, 16*16*sizeof(hip_bfloat16));
  hipMalloc(&d_c, 16*16*sizeof(float));

  FILE *outfile = stdout;
  bool pass;

  printheader(outfile, "A. Support for subnormal numbers");// ;

  printitem(outfile, "*) Bfloat16 subnormals in input (binary32 mode)");
  host_reset(h_a, h_b, h_c);
  h_a[0] = minsubnormal16;
  h_b[0] = __float2bfloat16(ldexp(1, 7));
  wmma_init_run(h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  printpass(outfile, h_c[0]==ldexp(1., -126));

  printitem(outfile, "*) Binary32 subnormals in input");
  host_reset(h_a, h_b, h_c);
  h_c[0] = minsubnormal32;
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  printpass(outfile, h_c[0] == minsubnormal32);

  printitem(outfile,
    "*) Bfloat16/binary32 subnormals in output (binary32 mode)");
  host_reset(h_a, h_b, h_c);
  h_a[0] = __float2bfloat16(ldexp(1., -126));
  h_b[0] = __float2bfloat16(ldexp(1., -1));
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  pass = h_c[0] == ldexp(1, -127);
  h_a[0] = __float2bfloat16(ldexp(1., -126));
  h_b[0] = one16;
  h_c[0] = ldexp(-1., -127);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  pass = pass && (h_c[0] == ldexp(1, -127));
  printpass(outfile, pass);

  printfooter(outfile);

  printheader(outfile, "B. Accuracy of the dot products ");// ;

  printitem(outfile, "*) Products are computed exactly ");
  host_reset(h_a, h_b, h_c);
  h_a[0] = belowone16;
  h_b[0] = belowone16;
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  pass = (h_c[0] == 1 - ldexp(1, -7) + ldexp(1, -16));
  size_t i,j;
  for (i=0; i<4; i++) {
    h_a[i] = belowone16;
    h_b[i] = belowone16;
  }
  h_c[0] = zero16;
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  pass = (h_c[0] == (4 * (1 - ldexp(1, -7) + ldexp(1, -16))));
  printpass(outfile, pass);

  printitem(outfile, "*) Products are accumulated in binary32 ");
  host_reset(h_a, h_b, h_c);
  pass = true;
  for (i=0; i<4; i++) {
    h_a[i] = 0.5;
    h_b[i] = __float2bfloat16(ldexp(1, -24));
  }
  h_c[0] = 1.;
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  pass = pass && h_c[0] == 1;
  printpass(outfile, pass);

  printitem(outfile, "*) Sum starts from largest element");
  host_reset(h_a, h_b, h_c);
  pass = true;
  for (i=0; i<4; i++) {
    h_a[i] = 0.5;
    h_b[i] = __float2bfloat16(ldexp(1, -24));
  }
  for (j=0; j<4; j++) {
    h_c[0] = ldexp(1, -24);
    if (j>0)
      h_a[j-1] = 0.5;
    h_b[j-1] = __float2bfloat16(ldexp(1, -24));
    h_a[j] = one16;
    h_b[j] = one16;
    wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
    pass = pass && h_c[0] == 1;
  }
  printpass(outfile, pass);

  printfooter(outfile);

  printheader(outfile, "C. Rounding modes in tensor core computations ");

  printitem(outfile, "*) Round-down for positive values ");
  host_reset(h_a, h_b, h_c);
  for (i=0; i<4; i++) {
    h_a[i] = one16;
  }
  h_b[0] = __float2bfloat16(2.);
  h_b[1] = __float2bfloat16(ldexp(1., -23) + ldexp(1., -24));
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  printpass(outfile, h_c[0] == 2.);

  printitem(outfile, "*) Round-up for negative values ");
  host_reset(h_a, h_b, h_c);
  for (i=0; i<4; i++) {
    h_a[i] = one16;
  }
  h_b[0] = __float2bfloat16(-2.);
  h_b[1] = __float2bfloat16(-ldexp(1., -23) - ldexp(1., -24));
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  printpass(outfile, h_c[0] == -2.);

  printfooter(outfile);

  printheader(outfile, "D. Features of the accumulator");

  printitem(outfile, "1) Extra bits in the significand alignment");
  host_reset(h_a, h_b, h_c);
  h_a[0] = one16;
  h_b[0] = one16;
  h_c[0] = -belowone;
  // h_c[0] = 1.0f;
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  assert(1 - belowone == ldexp(1., -24));
  assert(gapbelowone == ldexp(1., -24));
  printpass(outfile, h_c[0] == ldexp(1., -24));
  // fprintf(outfile, "%x \n", ((uint32_t*)h_c)[0]);

  printitem(outfile, "2) Normalization in addition");
  host_reset(h_a, h_b, h_c);
  for (i=0; i<4; i++) {
    h_a[i] = one16;
    h_b[i] = __float2bfloat16(ldexp(1, -24));
  }
  h_c[0] = 1. - ldexp(1., -24);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  pass = h_c[0] == 1. + ldexp(1., -23);
  printpass(outfile, pass);

  printitem(outfile, "3) Normalization in subtraction");
  host_reset(h_a, h_b, h_c);
  h_a[0] = one16;
  h_a[1] = one16;
  h_b[0] = one16;
  h_b[1] = __float2bfloat16(-ldexp(1., -24));
  h_c[0] = -1. + ldexp(1., -24);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  pass = pass && h_c[0] == 0.0;
  printpass(outfile, pass);

  printitem(outfile, "4) Extra bits for carry out");
  host_reset(h_a, h_b, h_c);
  for (i=0; i<4; i++) {
    h_a[i] = one16;
    h_b[i] = one16;
  }
  pass = true;
  for (i=0; i<4; i++) {
    if (i>0)
      h_b[i-1] = one16;
    h_b[i] = __float2bfloat16(ldexp(1., -23));
    h_c[0] = 1. + ldexp(1., -22) + ldexp(1., -23);
    wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
    pass = pass && h_c[0] == 4. + ldexp(1., -21);
  }

  // Test for the third bit
  host_reset(h_a, h_b, h_c);
  for (i=0; i<4; i++) {
    h_a[i] = one16;
    h_b[i] = one16;
  }
  pass = true;
  h_b[0] = one16;
  h_b[1] = __float2bfloat16(1.5);
  h_b[2] = __float2bfloat16(1.75);
  h_b[3] = __float2bfloat16(1.875);
  h_c[0] = 1.875;
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  pass = pass && h_c[0] == 8.;

  // Round-down in normalization of positive values.
  host_reset(h_a, h_b, h_c);
  for (i=0; i<4; i++) {
    h_a[i] = one16;
    h_b[i] = one16;
  }
  h_c[0] = 1. + ldexp(1., -22) + ldexp(1., -23);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  pass = pass && (h_c[0] == 5.);

  // Round-up in normalization of negative values.
  host_reset(h_a, h_b, h_c);
  for (i=0; i<4; i++) {
    h_a[i] = one16;
    h_b[i] = minusone16;
  }
  h_c[0] = -1. - ldexp(1., -22) - ldexp(1., -23);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  pass = pass && (h_c[0] == -5.);

  printpass(outfile, pass);

  printitem(outfile, "5) Monotonicity of dot product");
  host_reset(h_a, h_b, h_c);
  for (i=0; i<3; i++) {
    h_a[i] = 0.5;
    h_b[i] = ldexp(1., -24);
  }
  for (i=3; i<4; i++) {
    h_a[i] = 0.5;
    h_b[i] = ldexp(1., -23)+ldexp(1., -24);
  }
  h_c[0] = 1. - ldexp(1., -24);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  float partial = h_c[0];
  h_c[0] = 1.0;
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  printpass(outfile, h_c[0] < partial);

  printfooter(outfile);

  // Free dynamically allocated memory.
  //  free(h_a);
  //  free(h_b);
  free(h16_c);
  hipFree(d16_a);
  hipFree(d16_b);
  hipFree(d16_c);
  hipFree(d_c);
  free(h_c);
}
