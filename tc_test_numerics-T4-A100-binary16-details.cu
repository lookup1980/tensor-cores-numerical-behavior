/*
 * Copyright (c) 2020, Massimiliano Fasi and Mantas Mikaitis
 *
 * This program is free software: you can redistribute it and/or modify it under
 * the terms of the GNU General Public License as published by the Free Software
 * Foundation, version 2.
 *
 * This program is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS
 * FOR A PARTICULAR PURPOSE. See the GNU General Public License for more
 * details.
 *
 *  You should have received a copy of the GNU General Public License along with
 *  this program. If not, see <http://www.gnu.org/licenses/>.
 */


#include <hip/hip_runtime.h>
#include <assert.h>
#include <unistd.h>
#include <cstdint>
#include <chrono>
#include <iostream>
#include <mma.h>
#include <iomanip>

using namespace nvcuda;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*******************
 * Debug functions *
 *******************/
/* Print the elements of the m x n matrix A. The elements are assumed to be
   stored by columns if `bycols` is `true` and by rows if `bycols` is false. */
template <typename floattype>
void print_matrix (half *a,
                   size_t m, size_t n,
                   bool bycols) {
  int i, j;
  if (bycols) {
    for (i=0; i<m; i++) {
      for (j=0; j<n; j++)
        std::cout << __half2float(a[j*n+i]) << " ";
      std::cout << std::endl;
    }
    std::cout << std::endl;
  } else {
    for (i=0; i<m; i++ ) {
      for (j=0; j<n; j++)
        std::cout << __half2float(a[i*m+j]) << " ";
      std::cout  << std::endl;
    }
    std::cout << std::endl;
   }
}


/****************************************************
 * Memory management and wmma::mma_sync() interface *
 ****************************************************/

/* Set the entries of host arrays to zero. */
template <typename returntype>
void host_reset(half *a, half *b, returntype *c) {
  memset(a, 0, 16*16*sizeof(half));
  memset(b, 0, 16*16*sizeof(half));
  memset(c, 0, 16*16*sizeof(returntype));
}

/* Compute C += A*B, where A, B, and C are 16x16x16 matrices.
   The matrix C is initialized to 0 when `init` is true. */
template <typename returntype>
__global__ void wmma_ker(half *a, half *b, returntype *c, bool init) {

  // Declare fragments.
  wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_fragment;
  wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> b_fragment;
  wmma::fragment<wmma::accumulator, 16, 16, 16, returntype> c_fragment;

  // Load input matrices and initialize output (if required).
  wmma::load_matrix_sync(a_fragment, a, 16);
  wmma::load_matrix_sync(b_fragment, b, 16);
  if (init)
    wmma::fill_fragment(c_fragment, 0.0f);
  else
    wmma::load_matrix_sync(c_fragment, c, 16, wmma::mem_col_major);

  // Multiply
  wmma::mma_sync(c_fragment, a_fragment, b_fragment, c_fragment);

  // Store the output
  wmma::store_matrix_sync(c, c_fragment, 16, wmma::mem_col_major);
}

/* Copy data from host to device, perform the operation, and copy result back to
   host. */
template <typename returntype>
void wmma_init_run (half *h_a, half *h_b, returntype *h_c,
                    half *d_a, half *d_b, returntype *d_c,
                    bool init) {

  // Copy input from host to device.
  hipMemcpy(d_a, h_a, 16*16*sizeof(half), hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, 16*16*sizeof(half), hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, 16*16*sizeof(returntype), hipMemcpyHostToDevice);

  // Perform matrix multiplication.
  wmma_ker<<<1,32>>>(d_a, d_b, d_c, init);

  // Copy result from device to host.
  hipMemcpy(h_c, d_c, 16*16*sizeof(returntype), hipMemcpyDeviceToHost);
}


// mgu
void printhalf(half h) {
  uint16_t *ph = (uint16_t*) &h;
  printf("0x%04hx ", *ph);
}
void printfloat(float h) {
  uint32_t *ph = (uint32_t*) &h;
  printf("0x%08x ", *ph);
}

void float_to_half(float *fa, float *fb, half *ha, half *hb) {
  for (size_t i = 0; i < 4; i++)
  {
    ha[i] = __float2half(fa[i]);
    hb[i] = __float2half(fb[i]);
  }
}

float get_float_reference(float *fa, float *fb, float c) {
  float cc = 0.0f;
  for (size_t i = 0; i < 4; i++)
  {
    cc += fa[i] * fb[i];
  }
  return cc + c;
}

void print_result(float *fa, float *fb, half *ha, half *hb, float c, float result) {
  printf("\nh_a: ");
  for (size_t i = 0; i < 4; i++)
  {
    printf("%04hx ", *(uint16_t*)&ha[i]);
  }
  printf("\nh_b: ");
  for (size_t i = 0; i < 4; i++)
  {
    printf("%04hx ", *(uint16_t*)&hb[i]);
  }
  printf("\nf_c: ");
  printf("%08x ", *(uint32_t*)&c);
  printf("\nr  : ");
  printf("%08x ", *(uint32_t*)&result);

  float ref = get_float_reference(fa, fb, c);
  printf("\nref: ");
  printf("%08x ", *(uint32_t*)&ref);

  printf("\n");
}

void reset_all(float *fa, float *fb, half *ha, half *hb, float *fc) {
  memset(fa, 0, 16*16*sizeof(float));
  memset(fb, 0, 16*16*sizeof(float));
  memset(ha, 0, 16*16*sizeof(half));
  memset(hb, 0, 16*16*sizeof(half));
  memset(fc, 0, 16*16*sizeof(float));
}

void my_test_addr() {

  // Declare pointers and allocate memory.
  half *h_a, *h_b, *h16_c, *d16_a, *d16_b, *d16_c;
  float *d_c, *h_c;

  h_a = new half[16*16];
  h_b = new half[16*16];
  h_c = new float[16*16];
  h16_c = new half[16*16];

  hipMalloc(&d16_a, 16*16*sizeof(half));
  hipMalloc(&d16_b, 16*16*sizeof(half));
  hipMalloc(&d16_c, 16*16*sizeof(half));
  hipMalloc(&d_c, 16*16*sizeof(float));

  // mgu
  float fa[16*16] = {};
  float fb[16*16] = {};
  float temp = 0;

  printf("\n");
  printf("---------------------------------------------------\n");
  printf("Tests for addr of DP\n");

  // case
  printf("\n");
  printf("case: 2^-23 + 1");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = ldexp(1.f, -23);
  fb[0] = 1.0f;
  fa[3] = 1.0f;
  fb[3] = 1.0f;
  h_c[0] = 0.0f;
  temp = h_c[0];
  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 2^-24 + 1");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = ldexp(1.f, -24);
  fb[0] = 1.0f;
  fa[3] = 1.0f;
  fb[3] = 1.0f;
  h_c[0] = 0.0f;
  temp = h_c[0];

  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 2^-12 * 2^-12 + 1");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = ldexp(1.f, -12);
  fb[0] = ldexp(1.f, -12);
  fa[3] = 1.0f;
  fb[3] = 1.0f;
  h_c[0] = 0.0f;
  temp = h_c[0];

  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 2^-23 - 1 + 1");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = ldexp(1.f, -23);
  fb[0] = 1.0f;
  fa[1] = -1.0f;
  fb[1] = 1.0f;
  fa[3] = 1.0f;
  fb[3] = 1.0f;
  h_c[0] = 0.0f;
  temp = h_c[0];
  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 2^-24 - 1 + 1");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = ldexp(1.f, -24);
  fb[0] = 1.0f;
  fa[1] = -1.0f;
  fb[1] = 1.0f;
  fa[3] = 1.0f;
  fb[3] = 1.0f;
  h_c[0] = 0.0f;
  temp = h_c[0];
  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // // case
  // printf("\n");
  // printf("case: 2^-24 - 2 + 2");
  // reset_all(fa, fb, h_a, h_b, h_c);
  // fa[0] = ldexp(1.f, -24);
  // fb[0] = 1.0f;
  // fa[2] = -1.0f;
  // fb[2] = 2.0f;
  // fa[3] = 1.0f;
  // fb[3] = 2.0f;
  // h_c[0] = 0.0f;
  // temp = h_c[0];
  // float_to_half(fa, fb, h_a, h_b);
  // wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  // print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // // case
  // reset_all(fa, fb, h_a, h_b, h_c);
  // fa[0] = ldexp(1.f, -24);
  // fb[0] = 1.0f;
  // fa[1] = -1.0f;
  // fb[1] = 2.0f;
  // fa[3] = 1.0f;
  // fb[3] = 2.0f;
  // h_c[0] = 0.0f;
  // temp = h_c[0];
  // float_to_half(fa, fb, h_a, h_b);
  // wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  // print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 2 - 2 + 2^-24");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = 1.0f;
  fb[0] = 2.0f;
  fa[2] = -1.0f;
  fb[2] = 2.0f;
  fa[3] = ldexp(1.f, -24);
  fb[3] = 1.0f;
  h_c[0] = 0.0f;
  temp = h_c[0];
  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // // case
  // printf("\n");
  // printf("case: 2^-24 - 4 + 4");
  // reset_all(fa, fb, h_a, h_b, h_c);
  // fa[0] = ldexp(1.f, -24);
  // fb[0] = 1.0f;
  // fa[2] = -1.0f;
  // fb[2] = 4.0f;
  // fa[3] = 1.0f;
  // fb[3] = 4.0f;
  // h_c[0] = 0.0f;
  // temp = h_c[0];
  // float_to_half(fa, fb, h_a, h_b);
  // wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  // print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // // case
  // reset_all(fa, fb, h_a, h_b, h_c);
  // fa[0] = ldexp(1.f, -24);
  // fb[0] = 1.0f;
  // fa[1] = -1.0f;
  // fb[1] = 4.0f;
  // fa[3] = 1.0f;
  // fb[3] = 4.0f;
  // h_c[0] = 0.0f;
  // temp = h_c[0];
  // float_to_half(fa, fb, h_a, h_b);
  // wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  // print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 2^4 - 2^4 + 2^-24");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = 1.0f;
  fb[0] = ldexp(1.f, 4);
  fa[2] = -1.0f;
  fb[2] = ldexp(1.f, 4);
  fa[3] = ldexp(1.f, -24);
  fb[3] = 1.0f;
  h_c[0] = 0.0f;
  temp = h_c[0];
  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 2^4 - 2^4 + 2^-23");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = 1.0f;
  fb[0] = ldexp(1.f, 4);
  fa[2] = -1.0f;
  fb[2] = ldexp(1.f, 4);
  fa[3] = ldexp(1.f, -23);
  fb[3] = 1.0f;
  h_c[0] = 0.0f;
  temp = h_c[0];
  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 2^4 - 2^4 + 2^-22");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = 1.0f;
  fb[0] = ldexp(1.f, 4);
  fa[2] = -1.0f;
  fb[2] = ldexp(1.f, 4);
  fa[3] = ldexp(1.f, -22);
  fb[3] = 1.0f;
  h_c[0] = 0.0f;
  temp = h_c[0];
  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 2^4 - 2^4 + 2^-21");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = 1.0f;
  fb[0] = ldexp(1.f, 4);
  fa[2] = -1.0f;
  fb[2] = ldexp(1.f, 4);
  fa[3] = ldexp(1.f, -21);
  fb[3] = 1.0f;
  h_c[0] = 0.0f;
  temp = h_c[0];
  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 2^4 - 2^4 + 2^-20");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = 1.0f;
  fb[0] = ldexp(1.f, 4);
  fa[2] = -1.0f;
  fb[2] = ldexp(1.f, 4);
  fa[3] = ldexp(1.f, -20);
  fb[3] = 1.0f;
  h_c[0] = 0.0f;
  temp = h_c[0];
  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 2^16 - 2^16 + 2^-24");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = ldexp(1.f, 8);
  fb[0] = ldexp(1.f, 8);
  fa[2] = -ldexp(1.f, 8);
  fb[2] = ldexp(1.f, 8);
  fa[3] = ldexp(1.f, -24);
  fb[3] = 1.0f;
  h_c[0] = 0.0f;
  temp = h_c[0];
  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 2^24 - 2^24 + 2^-24");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = ldexp(1.f, 12);
  fb[0] = ldexp(1.f, 12);
  fa[2] = -ldexp(1.f, 12);
  fb[2] = ldexp(1.f, 12);
  fa[3] = ldexp(1.f, -24);
  fb[3] = 1.0f;
  h_c[0] = 0.0f;
  temp = h_c[0];
  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 2^29 - 2^29 + 2^-24");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = ldexp(1.f, 14);
  fb[0] = ldexp(1.f, 15);
  fa[2] = -ldexp(1.f, 14);
  fb[2] = ldexp(1.f, 15);
  fa[3] = ldexp(1.f, -24);
  fb[3] = 1.0f;
  h_c[0] = 0.0f;
  temp = h_c[0];
  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);


  // Free dynamically allocated memory.
  free(h_a);
  free(h_b);
  free(h16_c);
  hipFree(d16_a);
  hipFree(d16_b);
  hipFree(d16_c);
  hipFree(d_c);
  free(h_c);
}

void my_test_normalize() {

  // Declare pointers and allocate memory.
  half *h_a, *h_b, *h16_c, *d16_a, *d16_b, *d16_c;
  float *d_c, *h_c;

  h_a = new half[16*16];
  h_b = new half[16*16];
  h_c = new float[16*16];
  h16_c = new half[16*16];

  hipMalloc(&d16_a, 16*16*sizeof(half));
  hipMalloc(&d16_b, 16*16*sizeof(half));
  hipMalloc(&d16_c, 16*16*sizeof(half));
  hipMalloc(&d_c, 16*16*sizeof(float));

  // mgu
  float fa[16*16] = {};
  float fb[16*16] = {};
  float temp = 0;

  printf("\n");
  printf("---------------------------------------------------\n");
  printf("Tests for normalization before add accumulator\n");

  // case
  printf("\n");
  printf("case: 1, 2^-23");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = 1.0f;
  fb[0] = 1.0f;
  fa[1] = -1.0f;
  fb[1] = 1.0f;
  h_c[0] = ldexp(1.f, -23);
  temp = h_c[0];

  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 1, 2^-24");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = 1.0f;
  fb[0] = 1.0f;
  fa[1] = -1.0f;
  fb[1] = 1.0f;
  h_c[0] = ldexp(1.f, -24);
  temp = h_c[0];

  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 1, 2^-25");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = 1.0f;
  fb[0] = 1.0f;
  fa[1] = -1.0f;
  fb[1] = 1.0f;
  h_c[0] = ldexp(1.f, -25);
  temp = h_c[0];

  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 1, 2^-26");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = 1.0f;
  fb[0] = 1.0f;
  fa[1] = -1.0f;
  fb[1] = 1.0f;
  h_c[0] = ldexp(1.f, -26);
  temp = h_c[0];

  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);

  // case
  printf("\n");
  printf("case: 1, 2^-40");
  reset_all(fa, fb, h_a, h_b, h_c);
  fa[0] = 1.0f;
  fb[0] = 1.0f;
  fa[1] = -1.0f;
  fb[1] = 1.0f;
  h_c[0] = ldexp(1.f, -40);
  temp = h_c[0];

  float_to_half(fa, fb, h_a, h_b);
  wmma_init_run (h_a, h_b, h_c, d16_a, d16_b, d_c, false);
  print_result(fa, fb, h_a, h_b, temp, h_c[0]);


  // Free dynamically allocated memory.
  free(h_a);
  free(h_b);
  free(h16_c);
  hipFree(d16_a);
  hipFree(d16_b);
  hipFree(d16_c);
  hipFree(d_c);
  free(h_c);
}

/***************
 * EXPERIMENTS *
 ***************/
int main(int argc, char** argv){
  my_test_addr();
  my_test_normalize();
}
